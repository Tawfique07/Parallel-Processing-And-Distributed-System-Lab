#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <hip/hip_runtime.h>

struct Contact {
    char name[256];
    char phoneNumber[256];
};

__device__ bool deviceStrStr(const char* haystack, const char* needle, int needleLength) {
    for (int i = 0; i < needleLength; ++i) {
        if (haystack[i] != needle[i]) {
            return false;
        }
    }
    return true;
}

__global__ void searchContactsKernel(Contact* contacts, int size, const char* searchName, int searchNameLength) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        if (deviceStrStr(contacts[tid].name, searchName, searchNameLength)) {
            printf("Name: %s, Phone: %s\n", contacts[tid].name, contacts[tid].phoneNumber);
        }
    }
}

void printPhonebook(const std::vector<Contact>& phonebook) {
    std::cout << "Phonebook:\n";
    for (const auto& contact : phonebook) {
        std::cout << "Name: " << contact.name << ", Phone: " << contact.phoneNumber << "\n";
    }
    std::cout << "\n";
}

void searchContactsCUDA(const Contact* h_phonebook, int size, const char* searchName) {
    size_t contactsSize = size * sizeof(Contact);

    // Allocate device memory for contacts
    Contact* d_contacts;
    hipMalloc((void**)&d_contacts, contactsSize);

    // Copy contacts data from host to device
    hipMemcpy(d_contacts, h_phonebook, contactsSize, hipMemcpyHostToDevice);

    // Allocate device memory for searchName
    int searchNameLength = strlen(searchName) + 1;
    char* d_searchName;
    hipMalloc((void**)&d_searchName, searchNameLength);

    // Copy searchName to device
    hipMemcpy(d_searchName, searchName, searchNameLength, hipMemcpyHostToDevice);

    // Set up grid and block dimensions
    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;

    // Launch the search kernel
    searchContactsKernel<<<gridSize, blockSize>>>(d_contacts, size, d_searchName, searchNameLength);

    // Synchronize to ensure kernel execution is complete
    hipDeviceSynchronize();

    // Free device memory
    hipFree(d_contacts);
    hipFree(d_searchName);
}

std::vector<Contact> readPhonebook(const std::string& filename) {
    std::vector<Contact> phonebook;
    std::ifstream file(filename);

    if (!file.is_open()) {
        std::cerr << "Error opening file: " << filename << std::endl;
        return phonebook;
    }

    Contact contact;
    while (file >> contact.name >> contact.phoneNumber) {
        phonebook.push_back(contact);
    }

    file.close();
    return phonebook;
}

int main() {
    const std::string phonebookFilename = "phonebook.txt";
    const char* searchName = "Charlie";  // Change this to the desired search name

    // Read phonebook from file
    std::vector<Contact> phonebook = readPhonebook(phonebookFilename);

    if (!phonebook.empty()) {
        // Print the entire phonebook
        printPhonebook(phonebook);

        // Search contacts using CUDA
        searchContactsCUDA(phonebook.data(), phonebook.size(), searchName);
    }

    return 0;
}